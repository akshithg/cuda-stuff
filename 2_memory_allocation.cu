// Memory Allocation
// Simple op to demonstrate memory allocation
// malloc, free, memcpy = cudaMalloc, cudaFree, cudaMemcpy


#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void add(int *a, int *b, int *c) {
    *c = *a + *b;
}

int main(void) {
    int a, b, c; // host copy
    int *d_a, *d_b, *d_c; // device copy
    int size = sizeof(int);

    // allocate mem for device copies
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    a = 1;
    b = 5;

    // copy inputs to device
    // cudaMemcpy(destination, source, size, direction);
    hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

    // launch add() on GPU
    add<<<1, 1>>>(d_a, d_b, d_c);

    // copy result back to host
    hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

    // cleanup
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    printf("%d + %d = %d\n", a, b, c);

    return 0;
}
