// Blocks
// blockIdx -> block index


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#define N 2048*2048

__global__ void add(int *a, int *b, int *c) {
    // use blockIdx.x to access block index
    c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

void random_ints(int *a, int n){
   int i;
   for (i = 0; i < n; ++i)
        a[i] = rand()%100;
}

int main(void) {
    int *a, *b, *c; // host copy
    int *d_a, *d_b, *d_c; // device copy
    int size = N * sizeof(int);

    // allocate mem for device copies
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    a = (int *)malloc(size); random_ints(a, N);
    b = (int *)malloc(size); random_ints(b, N);
    c = (int *)malloc(size);

    clock_t start, end;
    double cpu_time_used;
    start = clock();

    // copy inputs to device
    // cudaMemcpy(destination, source, size, direction);
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // launch add() on GPU
    // instead of executing add once, execute N time in parallel
    add<<<N, 1>>>(d_a, d_b, d_c);

    // copy result back to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    end = clock();
    cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
    printf("exec time: %f seconds\n", cpu_time_used);

    // cleanup
    free(a); free(b); free(c);
    hipFree(d_a); hipFree(d_b); hipFree(d_c);

    return 0;
}
