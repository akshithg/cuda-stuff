// Threads
// threadIdx -> thread index


#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 512

__global__ void add(int *a, int *b, int *c) {
    // use threadIdx.x to access thread index
    c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

void random_ints(int *a, int n){
   int i;
   for (i = 0; i < n; ++i)
        a[i] = rand()%100;
}

int main(void) {
    int *a, *b, *c; // host copy
    int *d_a, *d_b, *d_c; // device copy
    int size = N * sizeof(int);
    int i;

    // allocate mem for device copies
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    a = (int *)malloc(size); random_ints(a, N);
    b = (int *)malloc(size); random_ints(b, N);
    c = (int *)malloc(size);

    // copy inputs to device
    // cudaMemcpy(destination, source, size, direction);
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // launch add() kernel on GPU with N threads
    add<<<1, N>>>(d_a, d_b, d_c);

    // copy result back to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    for(i=0; i<N; i++) {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    // cleanup
    free(a); free(b); free(c);
    hipFree(d_a); hipFree(d_b); hipFree(d_c);

    return 0;
}
